#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <hip/hip_vector_types.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions

texture<uchar4, 2, hipReadModeNormalizedFloat> rgbaTex;
texture<float, 2, hipReadModeNormalizedFloat> grayTex;



__device__ uint rgbaFloatToInt(float4 rgba)
{
	rgba.x = __saturatef(fabs(rgba.x));   // clamp to [0.0, 1.0]
	rgba.y = __saturatef(fabs(rgba.y));
	rgba.z = __saturatef(fabs(rgba.z));
	rgba.w = __saturatef(fabs(rgba.w));
	return (uint(rgba.w * 255.0f) << 24) | (uint(rgba.z * 255.0f) << 16) | (uint(rgba.y * 255.0f) << 8) | uint(rgba.x * 255.0f);
}


__device__ float4 rgbaIntToFloat(uint c)
{
	float4 rgba;
	rgba.x = (c & 0xff) * 0.003921568627f;       //  /255.0f;
	rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;  //  /255.0f;
	rgba.z = ((c >> 16) & 0xff) * 0.003921568627f; //  /255.0f;
	rgba.w = ((c >> 24) & 0xff) * 0.003921568627f; //  /255.0f;
	return rgba;
}


__global__ void
d_passthrough_texture(uint *od, int w, int h)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= w || y >= h)
	{
		return;
	}
	float4 pixel = tex2D(rgbaTex, x, y);
	float4 pixel_inverted;
	pixel_inverted.x = pixel.z;
	pixel_inverted.y = pixel.y;
	pixel_inverted.z = pixel.x;
	od[y * w + x] = rgbaFloatToInt(pixel_inverted);
	return;
}




extern "C"
void passthrough_texture(unsigned int *dOutputImage, unsigned int *dInputImage, int width, int height, int pitch)
{
	// Bind the array to the texture
	hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
	checkCudaErrors(hipBindTexture2D(0, rgbaTex, dInputImage, desc, width, height, pitch));

	const dim3 threads_per_block(16, 16);
	dim3 num_blocks;
	num_blocks.x = (width + threads_per_block.x - 1) / threads_per_block.x;
	num_blocks.y = (height + threads_per_block.y - 1) / threads_per_block.y;
	
	d_passthrough_texture << <  num_blocks, threads_per_block >> >(dOutputImage, width, height);
}

