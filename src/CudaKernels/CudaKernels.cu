#include "hip/hip_runtime.h"
#include "CudaKernels.h"

#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/device_vector.h>

extern "C"
{

	__global__ void update_vb(float *verts, int vertex_count, float timeElapsed)
	{
		const unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;

		if (threadId < vertex_count * 4)
		{
			float valx = verts[threadId * 4 + 0];
			float valy = verts[threadId * 4 + 1];
			float valz = verts[threadId * 4 + 2];


			verts[threadId * 4 + 0] = valx * timeElapsed;
			verts[threadId * 4 + 1] = valy * timeElapsed;
			verts[threadId * 4 + 2] = valz * timeElapsed;
		}
	}

	void cuda_kernel(float *verts, int vertex_count, float timeElapsed)
	{
		if (vertex_count > 1024)
			update_vb << <vertex_count / 1024 + 1, 1024 >> >(verts, vertex_count, timeElapsed);
		else
			update_vb << <1, vertex_count >> >(verts, vertex_count, timeElapsed);
	}

};